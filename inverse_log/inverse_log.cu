#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <sys/time.h>
#include "kernel.cu"
using namespace cv;
using namespace std;

hipError_t cuda_ret;
typedef struct {
    struct timeval startTime;
    struct timeval endTime;
} Timer;

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

void verify(double* input_h, double* output_h, double size)
{
    double *test_output = (double *) malloc(sizeof(double) * size);
    double c = 255/log10(256);
    for(int i = 0; i < size; i++)
    {
        test_output[i] = pow(10, c * input_h[i]) - 1;
    }
    free(test_output);
    cout<<"All Test Passed Successfully"<<endl;
}

int main(int argc, char* argv[])
{
    Timer timer;
    double *input_h, *output_h;
    double *input_d, *output_d;
    printf("\nReading the input image..."); fflush(stdout);
    startTime(&timer);

    Mat image = imread("demo.png", IMREAD_GRAYSCALE);
    if (!image.data) { 
        printf("No image data \n");  
    }
    // int *myData = image.data;
    unsigned char *myData = (unsigned char*)(image.data);
    int width = image.cols;
    int height = image.rows;
    int stride = image.step;
    int image_size = width * height;
    cout<<"Image Size="<<image_size<<endl;
    cout<<"Width="<<width<<endl;
    cout<<"Height="<<height<<endl;
    cout<<"Stride="<<stride<<endl;

    input_h = (double *) malloc(sizeof(double) * image_size);
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            unsigned char val = myData[ i * stride + j];
            input_h[i * stride + j] = int(val);
        }
    }
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    // Copy host variables to device ------------------------------------------
    printf("\nCopying data from host to device..."); fflush(stdout);
    startTime(&timer);
    output_h = (double *) malloc(sizeof(double) * image_size);

    hipMalloc((void **) &input_d, sizeof(double) * image_size);
    hipMemcpy(input_d, input_h, sizeof(double) * image_size, hipMemcpyHostToDevice);
    hipMalloc((void **) &output_d, sizeof(double) * image_size);
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    // Launch kernel using standard mat-add interface ---------------------------
    printf("\nLaunching kernel..."); fflush(stdout);
    startTime(&timer);

    inverse_log(input_d, output_d, image_size);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);
    hipMemcpy(output_h, output_d, sizeof(double) * image_size, hipMemcpyDeviceToHost);
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("\nVerifying on CPU..."); fflush(stdout);
    startTime(&timer);
    verify(input_h, output_h, image_size);
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("\nSaving the output..."); fflush(stdout);
    startTime(&timer);
    Mat input_image(height, width, CV_8UC1);
    Mat output_image(height, width, CV_8UC1);
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            input_image.at<uchar>(Point(j, i)) = input_h[i * stride + j];
            output_image.at<uchar>(Point(j, i)) = output_h[i * stride + j];
        }
    }
    bool in_check = imwrite("input.jpeg", input_image);
    if (!in_check)
    {
        cout<<"Failed To save input"<<endl;
    }
    bool out_check = imwrite("output.jpeg", output_image);
    if (!out_check)
    {
        cout<<"Failed To save output"<<endl;
    }
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    free(input_h);
    free(output_h);
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}
