#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <ctime>
#include "kernel.cu"
using namespace cv;
using namespace std;

// void verify(double* input_h, double* output_h, double size)
// {
//     double *test_output = (double *) malloc(sizeof(double) * size);
//     double c = 255/log10(256);
//     for(int i = 0; i < size; i++)
//     {
//         test_output[i] = c * log(1 + input[i])
//     }
//     int count = 0;
//     for(int i = 0; i < size; i++)
//     {
//         if(test_output[i] != output_h[i])
//         {
//             cout<<"Difference in value - "<<i<<" - "<<test_output[i]<<" - "<<output_h[i]<<endl;
//             count = count + 1;
//         }
//     }
//     free(test_output);
//     if (count == 0)
//     {
//         cout<<"All Test Passed Successfully"<<endl;
//     }
// }

int main(int argc, char* argv[])
{
    hipError_t cuda_ret;
    double *input_h, *output_h;
    double *input_d, *output_d;
    printf("\nReading the input image..."); fflush(stdout);

    Mat image = imread("demo.png", IMREAD_GRAYSCALE);
    if (!image.data) { 
        printf("No image data \n");  
    }
    // int *myData = image.data;
    unsigned char *myData = (unsigned char*)(image.data);
    int width = image.cols;
    int height = image.rows;
    int stride = image.step;
    int image_size = width * height;
    cout<<"Image Size="<<image_size<<endl;
    cout<<"Width="<<width<<endl;
    cout<<"Height="<<height<<endl;
    cout<<"Stride="<<stride<<endl;

    input_h = (double *) malloc(sizeof(double) * image_size);
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            unsigned char val = myData[ i * stride + j];
            input_h[i * stride + j] = int(val);
        }
    }
    // Copy host variables to device ------------------------------------------
    printf("\nCopying data from host to device..."); fflush(stdout);
    output_h = (double *) malloc(sizeof(double) * image_size);

    hipMalloc((void **) &input_d, sizeof(double) * image_size);
    hipMemcpy(input_d, input_h, sizeof(double) * image_size, hipMemcpyHostToDevice);
    hipMalloc((void **) &output_d, sizeof(double) * image_size);
    hipDeviceSynchronize();
    // Launch kernel using standard mat-add interface ---------------------------
    printf("\nLaunching kernel..."); fflush(stdout);
    time_t start = time(0);
    cout<<"Start Time - "<<start<<endl;

    inverse_log(input_d, output_d, image_size);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");
    time_t end = time(0);
    cout<<"End Time - "<<end<<endl;
    cout<<"\nTotal Time - "<<end-start<<endl;

    // Copy device variables from host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    hipMemcpy(output_h, output_d, sizeof(double) * image_size, hipMemcpyDeviceToHost);
    // verify(input_h, output_h, image_size);
    printf("\nSaving the output..."); fflush(stdout);
    Mat input_image(height, width, CV_8UC1);
    Mat output_image(height, width, CV_8UC1);
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            input_image.at<uchar>(Point(j, i)) = input_h[i * stride + j];
            output_image.at<uchar>(Point(j, i)) = output_h[i * stride + j];
        }
    }
    bool in_check = imwrite("input.jpeg", input_image);
    if (!in_check)
    {
        cout<<"Failed To save input"<<endl;
    }
    bool out_check = imwrite("output.jpeg", output_image);
    if (!out_check)
    {
        cout<<"Failed To save output"<<endl;
    }
    free(input_h);
    free(output_h);
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}
