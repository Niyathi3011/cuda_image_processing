#include <stdio.h>
#include <stdint.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>
#include "kernel.cu"
using namespace cv;
using namespace std;

void verify(unsigned int* input_h, unsigned int* output_h, unsigned int size)
{
    unsigned int *test_output = (unsigned int *) malloc(sizeof(unsigned int) * size);
    for(int i = 0; i < size; i++)
    {
        test_output[i] = 255 - input_h[i];
    }
    unsigned int count = 0;
    for(int i = 0; i < size; i++)
    {
        if(test_output[i] != output_h[i])
        {
            cout<<"Difference in value - "<<i<<" - "<<test_output[i]<<" - "<<output_h[i]<<endl;
            count = count + 1;
        }
    }
    free(test_output);
    if (count == 0)
    {
        cout<<"All Test Passed Successfully"<<endl;
    }
}

int main(int argc, char* argv[])
{
    hipError_t cuda_ret;
    unsigned int *input_h, *output_h;
    unsigned int *input_d, *output_d;

    Mat image = imread("demo.png", IMREAD_GRAYSCALE);
    if (!image.data) { 
        printf("No image data \n");  
    }
    uint8_t *myData = image.data;
    int width = image.cols;
    int height = image.rows;
    int _stride = image.step;
    unsigned int image_size = width * height;
    cout<<"Image Size="<<image_size<<endl;
    cout<<"Width="<<unsigned(width)<<endl;
    cout<<"Height="<<unsigned(height)<<endl;
    cout<<"Stride="<<unsigned(_stride)<<endl;

    input_h = (unsigned int *) malloc(sizeof(unsigned int) * image_size);
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            uint8_t val = myData[ i * _stride + j];
            input_h[i * _stride + j] = unsigned(val);
        }
    }
    output_h = (unsigned int *) malloc(sizeof(unsigned int) * image_size);

    hipMalloc((void **) &input_d, sizeof(unsigned int) * image_size);
    hipMemcpy(input_d, input_h, sizeof(unsigned int) * image_size, hipMemcpyHostToDevice);
    hipMalloc((void **) &output_d, sizeof(unsigned int) * image_size);
    hipDeviceSynchronize();

    inverted_image(input_d, output_d, image_size);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");

    // Copy device variables from host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    hipMemcpy(output_h, output_d, sizeof(unsigned int) * image_size, hipMemcpyDeviceToHost);
    verify(input_h, output_h, image_size);
    free(input_h);
    free(output_h);
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}
