#include "hip/hip_runtime.h"
#include <stdio.h>
# define BLOCK_SIZE 512
# define MAX_NUMBER_OF_BLOCK 16

__global__ void image_histogram_kernel(double* input, int size, double* histogram, int total_bins)
{
	
    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    __shared__ double local_ihisto[256];
    int i, stride;
    for ( i = threadIdx.x ; i < total_bins ; i += BLOCK_SIZE )
    {
        local_ihisto[i] = 0;
    }
    __syncthreads();

    i = threadIdx.x + blockIdx.x * blockDim.x;
    stride = blockDim.x * gridDim.x;
    int value;
    while ( i < size )
    {
        value = input[i]
        atomicAdd(&(local_ihisto[value]), 1);
        i += stride;
    }

    __syncthreads();
    for ( i = threadIdx.x ; i < total_bins ; i += BLOCK_SIZE )
    {
        atomicAdd(&(histogram[i]), local_ihisto[i]);
    }
	/*************************************************************************/
}


void image_histogram(double* input, int size, double* histogram, int total_bins) {

	  /*************************************************************************/
    //INSERT CODE HERE
    int totalBlocks = (size - 1)/BLOCK_SIZE + 1;
    if ( totalBlocks > MAX_NUMBER_OF_BLOCK )
    {
        totalBlocks = MAX_NUMBER_OF_BLOCK;
    }
    dim3 DimGrid(totalBlocks, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    image_histogram_kernel<<<DimGrid, DimBlock>>>(input, size, histogram, total_bins);
	  /*************************************************************************/
}