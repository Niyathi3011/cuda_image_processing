#include "hip/hip_runtime.h"
#include <stdio.h>
# define BLOCK_SIZE 512
# define MAX_NUMBER_OF_BLOCK 16

__global__ void image_histogram_kernel(unsigned int* input, unsigned int size, unsigned int* histogram, unsigned int total_bins)
{
	
    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    __shared__ unsigned int local_ihisto[total_bins];
    int i, stride;
    for ( i = threadIdx.x ; i < total_bins ; i += BLOCK_SIZE )
    {
        local_ihisto[i] = 0;
    }
    __syncthreads();

    i = threadIdx.x + blockIdx.x * blockDim.x;
    stride = blockDim.x * gridDim.x;
    while ( i < size )
    {
        atomicAdd(&(local_ihisto[input[i]]), 1);
        i += stride;
    }

    __syncthreads();
    for ( i = threadIdx.x ; i < total_bins ; i += BLOCK_SIZE )
    {
        atomicAdd(&(histogram[i]), local_ihisto[i]);
    }
	/*************************************************************************/
}


void image_histogram(unsigned int* input, unsigned int size, unsigned int* histogram, unsigned int total_bins) {

	  /*************************************************************************/
    //INSERT CODE HERE
    int totalBlocks = (size - 1)/BLOCK_SIZE + 1;
    if ( totalBlocks > MAX_NUMBER_OF_BLOCK )
    {
        totalBlocks = MAX_NUMBER_OF_BLOCK;
    }
    dim3 DimGrid(totalBlocks, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    image_histogram_kernel<<<DimGrid, DimBlock>>>(input, size, histogram, total_bins);
	  /*************************************************************************/
}