#include <stdio.h>
#include <stdint.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include "kernel.cu"
using namespace cv;
using namespace std;

hipError_t cuda_ret;
typedef struct {
    struct timeval startTime;
    struct timeval endTime;
} Timer;

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

void verify(double* input_h, double *final_output_h, int size, double* histogram_h, int total_bins)
{
    double *test_histogram = (double *) malloc(sizeof(double) * total_bins);
    double *test_cdf = (double *) malloc(sizeof(double) * total_bins);
    for(int i = 0; i < total_bins; i++)
    {
        test_histogram[i] = 0;
        test_cdf[i] = 0;
    }
    for(int i = 0; i < size; i++)
    {
        int v = input_h[i];
        test_histogram[v] = test_histogram[v] + 1;
    }
    int count = 0;
    for(int i = 0; i < total_bins; i++)
    {
        if(test_histogram[i] != histogram_h[i])
        {
            cout<<"Difference in value - "<<i<<" - "<<test_histogram[i]<<" - "<<histogram_h[i]<<endl;
            count = count + 1;
        }
    }
    if (count == 0)
    {
        cout<<"All Test Passed Successfully"<<endl;
    }
    for(int i = 0; i < total_bins; i++)
    {
        test_histogram[i] = test_histogram[i]/255;
    }
    double sum = 0;
    for(int i = 0; i < total_bins; i++)
    {
        sum = sum + test_histogram[i];
        test_cdf[i] = int(sum * 255);
    }
    double *final_output = (double *) malloc(sizeof(double) * size);
    for(int i = 0; i < size; i++)
    {
        int v = input_h[i];
        final_output[i] = test_cdf[v];
    }
    // count = 0;
    // for(int i = 0; i < size; i++)
    // {
    //     if(final_output[i] != final_output_h[i])
    //     {
    //         cout<<"Difference in value - "<<i<<" - "<<test_histogram[i]<<" - "<<histogram_h[i]<<endl;
    //         count = count + 1;
    //     }
    // }
    if (count == 0)
    {
        cout<<"All Test Passed Successfully"<<endl;
    }
    free(test_histogram);
}

int main(int argc, char* argv[])
{
    Timer timer;
    double *input_h, *histogram_h, *output_h, *cdf_h, *final_output_h, *ff;
    double *input_d, *histogram_d, *output_d, *cdf_d, *final_output_d;
    int total_bins = 256;
    printf("\nReading the input image..."); fflush(stdout);
    startTime(&timer);

    Mat image = imread("demo.png", IMREAD_GRAYSCALE);
    if (!image.data) { 
        printf("No image data \n");  
    }
    unsigned char *myData = (unsigned char*)(image.data);
    int width = image.cols;
    int height = image.rows;
    int stride = image.step;
    int image_size = width * height;
    cout<<"Image Size="<<image_size<<endl;
    cout<<"Width="<<unsigned(width)<<endl;
    cout<<"Height="<<unsigned(height)<<endl;
    cout<<"Stride="<<unsigned(stride)<<endl;

    input_h = (double*) malloc(sizeof(double) * image_size);
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            unsigned char val = myData[ i * stride + j];
            input_h[i * stride + j] = int(val);
        }
    }
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    // Copy host variables to device ------------------------------------------
    printf("\nCopying data from host to device..."); fflush(stdout);
    startTime(&timer);
    histogram_h = (double *) malloc(sizeof(double) * total_bins);
    output_h = (double *) malloc(sizeof(double) * total_bins);
    cdf_h = (double *) malloc(sizeof(double) * total_bins);
    final_output_h = (double *) malloc(sizeof(double) * image_size);
    ff = (double *) malloc(sizeof(double) * image_size);

    hipMalloc((void **) &input_d, sizeof(double) * image_size);
    hipMemcpy(input_d, input_h, sizeof(double) * image_size, hipMemcpyHostToDevice);
    hipMalloc((void **) &histogram_d, sizeof(double) * total_bins);
    hipMemset(histogram_d, 0, total_bins * sizeof(double));
    hipMalloc((void **) &output_d, sizeof(double) * total_bins);
    hipMalloc((void **) &cdf_d, sizeof(double) * total_bins);
    hipMalloc((void **) &final_output_d, sizeof(double) * image_size);
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    // Launch kernel using standard mat-add interface ---------------------------
    printf("\nLaunching kernel..."); fflush(stdout);
    startTime(&timer);

    image_histogram(input_d, image_size, histogram_d, output_d, cdf_d, final_output_d, total_bins);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    // Copy device variables from host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);
    hipMemcpy(histogram_h, histogram_d, sizeof(double) * total_bins, hipMemcpyDeviceToHost);
    hipMemcpy(output_h, output_d, sizeof(double) * total_bins, hipMemcpyDeviceToHost);
    hipMemcpy(cdf_h, cdf_d, sizeof(double) * total_bins, hipMemcpyDeviceToHost);
    hipMemcpy(final_output_h, final_output_d, sizeof(double) * image_size, hipMemcpyDeviceToHost);
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("\nVerifying on CPU..."); fflush(stdout);
    startTime(&timer);
    verify(input_h, final_output_h, image_size, histogram_h, total_bins);
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    // cout<<"\nImage Histogram Distribution\n"<<endl;
    // for(int i = 0; i < total_bins; i++)
    // {
    //     cout<<i<<" - "<<histogram_h[i]<<" - "<<output_h[i]<<" - "<<cdf_h[i]<<endl;
    // }
    for(int i = 0; i < image_size; i++)
    {
        int finalv = input_h[i];
        ff[i] = cdf_h[finalv];
    }
    printf("\nSaving the output..."); fflush(stdout);
    startTime(&timer);
    Mat input_image(height, width, CV_8UC1);
    Mat output_image(height, width, CV_8UC1);
    Mat ff_im(height, width, CV_8UC1);
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            input_image.at<uchar>(Point(j, i)) = input_h[i * stride + j];
            ff_im.at<uchar>(Point(j, i)) = final_output_h[i * stride + j];
            output_image.at<uchar>(Point(j, i)) = ff[i * stride + j];
        }
    }
    bool in_check = imwrite("input.jpeg", input_image);
    if (!in_check)
    {
        cout<<"Failed To save input"<<endl;
    }
    bool out_check = imwrite("output.jpeg", output_image);
    if (!out_check)
    {
        cout<<"Failed To save output"<<endl;
    }
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    // verify(input_h, image_size, histogram_h, total_bins);
    free(input_h);
    free(histogram_h);
    free(output_h);
    free(cdf_h);
    free(final_output_h);
    free(ff);
    hipFree(input_d);
    hipFree(histogram_d);
    hipFree(output_d);
    hipFree(cdf_d);
    hipFree(final_output_d);
    return 0;
}
