#include <stdio.h>
#include <stdint.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>
#include "kernel.cu"
using namespace cv;
using namespace std;

int main(int argc, char* argv[])
{
    hipError_t cuda_ret;
    double *input_h, *output_h;
    double *input_d, *output_d;

    Mat image = imread("demo.png", IMREAD_GRAYSCALE);
    if (!image.data) { 
        printf("No image data \n");  
    }
    unsigned char *myData = (unsigned char*)image.data;
    int width = image.cols;
    int height = image.rows;
    int _stride = image.step;
    unsigned int image_size = width * height;
    cout<<"Image Size="<<image_size<<endl;
    cout<<"Width="<<unsigned(width)<<endl;
    cout<<"Height="<<unsigned(height)<<endl;
    cout<<"Stride="<<unsigned(_stride)<<endl;

    input_h = (double *) malloc(sizeof(double) * image_size);
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            unsigned char val = myData[ i * _stride + j];
            input_h[i * _stride + j] = int(val);
        }
    }
    output_h = (double *) malloc(sizeof(double) * image_size);

    hipMalloc((void **) &input_d, sizeof(double) * image_size);
    hipMemcpy(input_d, input_h, sizeof(double) * image_size, hipMemcpyHostToDevice);
    hipMalloc((void **) &output_d, sizeof(double) * image_size);
    hipDeviceSynchronize();

    scaling(input_d, output_d, image_size);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");

    // Copy device variables from host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    hipMemcpy(output_h, output_d, sizeof(double) * image_size, hipMemcpyDeviceToHost);
    Mat input_image(height, width, CV_8UC1);
    Mat output_image(height, width, CV_8UC1);
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            input_image.at<uchar>(Point(j, i)) = input_h[i * stride + j];
            output_image.at<uchar>(Point(j, i)) = output_h[i * stride + j];
        }
    }
    bool in_check = imwrite("input.jpeg", input_image);
    if (!in_check)
    {
        cout<<"Failed To save input"<<endl;
    }
    bool out_check = imwrite("output.jpeg", output_image);
    if (!out_check)
    {
        cout<<"Failed To save output"<<endl;
    }
    free(input_h);
    free(output_h);
    hipFree(input_d);
    hipFree(output_d);
    return 0;
}