#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>
using namespace cv;
using namespace std;

# define BLOCK_SIZE 512


__global__ void scaling_kernel(unsigned int* input, unsigned int size)
{
	
    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < size)
    {
        input[i] = 2 * input[i];
    }
	/*************************************************************************/
}


void scaling(unsigned int* input, unsigned int size) {

	  /*************************************************************************/
    //INSERT CODE HERE
    dim3 DimGrid((size - 1)/BLOCK_SIZE + 1, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    scaling_kernel<<<DimGrid, DimBlock>>>(input, size);
	  /*************************************************************************/
}


int main(int argc, char* argv[])
{
    hipError_t cuda_ret;
    Mat image = imread("demo.png", IMREAD_GRAYSCALE);
    if (!image.data) { 
        printf("No image data \n");  
    }
    uint8_t *myData = image.data;
    int width = image.cols;
    int height = image.rows;
    int _stride = image.step;
    unsigned int size = width * height * sizeof(unsigned int);
    cout<<"Size="<<size<<endl;
    cout<<"Width="<<unsigned(width)<<endl;
    cout<<"Height="<<unsigned(height)<<endl;
    cout<<"Stride="<<unsigned(_stride)<<endl;
    unsigned int *image_vector = (unsigned int *) malloc(size);
    for(int i = 0; i < height; i++)
    {
        for(int j = 0; j < width; j++)
        {
            uint8_t val = myData[ i * _stride + j];
            image_vector[i * _stride + j] = unsigned(val);
        }
    }
    unsigned int *image_vector_d;
    hipMalloc((void **) &image_vector_d, size);
    unsigned int *o_image_vector = (unsigned int *) malloc(size);
    hipMemcpy(image_vector_d, image_vector, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    scaling(image_vector_d, size);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");

    // Copy device variables from host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    hipMemcpy(o_image_vector, image_vector_d, size, hipMemcpyDeviceToHost);
    cout<<o_image_vector[262143]<<" - "<<image_vector[262143]<<endl;
    free(image_vector);
    hipFree(o_image_vector);
    return 0;
}